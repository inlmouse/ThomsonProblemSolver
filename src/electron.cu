#include "hip/hip_runtime.h"
#include "../include/electron.hpp"
#include <iostream>

namespace thomson
{
#ifdef USE_CUDA
	template<>
	void electron<float>::normalize2shpere_gpu(hipblasHandle_t cublas_handle)
	{
		float norm;
		CUBLAS_CHECK(hipblasSnrm2(cublas_handle, dimension_, position_->gpu_data(), 1, &norm));
		norm = 1 / norm;
		CUBLAS_CHECK(hipblasSscal(cublas_handle, dimension_, &norm, position_->mutable_gpu_data(), 1));
	}

	template<>
	void electron<double>::normalize2shpere_gpu(hipblasHandle_t cublas_handle)
	{
		double norm;
		CUBLAS_CHECK(hipblasDnrm2(cublas_handle, dimension_, position_->gpu_data(), 1, &norm));
		norm = 1 / norm;
		CUBLAS_CHECK(hipblasDscal(cublas_handle, dimension_, &norm, position_->mutable_gpu_data(), 1));
	}

	template <typename Dtype>
	void electron<Dtype>::combineforce2zero_gpu()
	{
		CUDA_CHECK(hipMemset(combine_force_->mutable_gpu_data(), 0, sizeof(Dtype) * dimension_));
	}

	template<>
	void electron<float>::add1componentforce_gpu(hipblasHandle_t cublas_handle, const float* other_position)
	{
		tensor<float>* f_i = new tensor<float>(std::vector<int>{dimension_}, device_);
		CUDA_CHECK(hipMemcpy(f_i->mutable_gpu_data(), other_position, dimension_ * sizeof(float), hipMemcpyDeviceToDevice));
		float inverser = -1.0;
		CUBLAS_CHECK(hipblasSscal(cublas_handle, dimension_, &inverser, f_i->mutable_gpu_data(), 1));
		float alpha = 1.0;
		CUBLAS_CHECK(hipblasSaxpy(cublas_handle, dimension_, &alpha, position_->gpu_data(), 1, f_i->mutable_gpu_data(), 1));
		float f_norm;
		CUBLAS_CHECK(hipblasSnrm2(cublas_handle, dimension_, f_i->gpu_data(), 1, &f_norm));
		f_norm = 1 / (f_norm * f_norm);
		CUBLAS_CHECK(hipblasSscal(cublas_handle, dimension_, &f_norm, f_i->mutable_gpu_data(), 1));
		CUBLAS_CHECK(hipblasSaxpy(cublas_handle, dimension_, &alpha, f_i->gpu_data(), 1, combine_force_->mutable_gpu_data(), 1));
		delete f_i;
	}

	template<>
	void electron<double>::add1componentforce_gpu(hipblasHandle_t cublas_handle, const double* other_position)
	{
		double* temp = new double[3];
		tensor<double>* f_i = new tensor<double>(std::vector<int>{dimension_}, device_); 
		CUDA_CHECK(hipMemcpy(f_i->mutable_gpu_data(), other_position, dimension_ * sizeof(double), hipMemcpyDeviceToDevice));
		double inverser = -1.0;
		hipblasGetVector(dimension_, sizeof(double), f_i->gpu_data(), 1, temp, 1);
		CUBLAS_CHECK(hipblasDscal(cublas_handle, dimension_, &inverser, f_i->mutable_gpu_data(), 1));
		double alpha = 1.0;
		CUBLAS_CHECK(hipblasDaxpy(cublas_handle, dimension_, &alpha, position_->gpu_data(), 1, f_i->mutable_gpu_data(), 1));
		double f_norm;
		CUBLAS_CHECK(hipblasDnrm2(cublas_handle, dimension_, f_i->gpu_data(), 1, &f_norm));
		f_norm = 1 / (f_norm * f_norm);
		CUBLAS_CHECK(hipblasDscal(cublas_handle, dimension_, &f_norm, f_i->mutable_gpu_data(), 1));
		CUBLAS_CHECK(hipblasDaxpy(cublas_handle, dimension_, &alpha, f_i->gpu_data(), 1, combine_force_->mutable_gpu_data(), 1));
		delete f_i;
	}

	template<>
	float electron<float>::calculatedistance_gpu(hipblasHandle_t cublas_handle, const float* other_position)
	{
		tensor<float>* f_i = new tensor<float>(std::vector<int>{dimension_}, device_);
		CUDA_CHECK(hipMemcpy(f_i->mutable_gpu_data(), other_position, dimension_ * sizeof(float), hipMemcpyDefault));
		float inverser = -1.0;
		CUBLAS_CHECK(hipblasSscal(cublas_handle, dimension_, &inverser, f_i->mutable_gpu_data(), 1));
		float alpha = 1.0;
		CUBLAS_CHECK(hipblasSaxpy(cublas_handle, dimension_, &alpha, position_->gpu_data(), 1, f_i->mutable_gpu_data(), 1));
		float f_norm;
		CUBLAS_CHECK(hipblasSnrm2(cublas_handle, dimension_, f_i->gpu_data(), 1, &f_norm));
		delete f_i;
		return f_norm;
	}

	template<>
	double electron<double>::calculatedistance_gpu(hipblasHandle_t cublas_handle, const double* other_position)
	{
		tensor<double>* f_i = new tensor<double>(std::vector<int>{dimension_}, device_);
		CUDA_CHECK(hipMemcpy(f_i->mutable_gpu_data(), other_position, dimension_ * sizeof(double), hipMemcpyDefault));
		double inverser = -1.0;
		CUBLAS_CHECK(hipblasDscal(cublas_handle, dimension_, &inverser, f_i->mutable_gpu_data(), 1));
		double alpha = 1.0;
		CUBLAS_CHECK(hipblasDaxpy(cublas_handle, dimension_, &alpha, position_->gpu_data(), 1, f_i->mutable_gpu_data(), 1));
		double f_norm;
		CUBLAS_CHECK(hipblasDnrm2(cublas_handle, dimension_, f_i->gpu_data(), 1, &f_norm));
		delete f_i;
		return f_norm;
	}

	template <>
	void electron<float>::updateposition_gpu(hipblasHandle_t cublas_handle, float lr)
	{
		CUBLAS_CHECK(hipblasSaxpy(cublas_handle, dimension_, &lr, combine_force_->gpu_data(), 1, position_->mutable_gpu_data(), 1));
		normalize2shpere_gpu(cublas_handle);
		CUDA_CHECK(hipMemset(combine_force_->mutable_gpu_data(), 0, sizeof(float) * dimension_));
	}

	template <>
	void electron<double>::updateposition_gpu(hipblasHandle_t cublas_handle, double lr)
	{
		CUBLAS_CHECK(hipblasDaxpy(cublas_handle, dimension_, &lr, combine_force_->gpu_data(), 1, position_->mutable_gpu_data(), 1));
		normalize2shpere_gpu(cublas_handle);
		CUDA_CHECK(hipMemset(combine_force_->mutable_gpu_data(), 0, sizeof(double) * dimension_));
	}

	template class electron<float>;
	template class electron<double>;
#endif
}